#include "hip/hip_runtime.h"
#include <vector>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "im2ht_cuda.cuh"


at::Tensor
ht_cuda_forward(
            const at::Tensor &input,
            const at::Tensor &ht_index,
            const int height,
            const int width,
            const int ht_height,
            const int ht_width
        )
{

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(ht_index.is_contiguous(), "ht_index tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(ht_index.type().is_cuda(), "ht_index must be a CUDA tensor");

    const int batch = input.size(0);
    const int channel = input.size(1);
    const int height_ = input.size(2);
    const int width_ = input.size(3);

    const int num_votes = ht_index.size(0);
    const int num_elements = ht_index.size(1);
    AT_ASSERTM(num_elements == 3,
        "ht_index shape != [num_votes x 3]: (%d x %d).", num_votes, num_elements);
    
    AT_ASSERTM(height_ == height && width_== width,
        " image shape and given shape params do not match: (%d x %d vs %d x %d).", height_, width_, height, width);

    // AT_ASSERTM(ht_h <= ht_index.max().item(), 
    //     "ht_index larger than ht_h (%d vs %d).", ht_index.max().item(), ht_h);

    // output: [batch, channel,  ht_height, ht_width]
    auto output = at::zeros({batch * channel * ht_height * ht_width}, input.options());

    AT_DISPATCH_FLOATING_TYPES(input.type(), "im2ht_cuda_forward", ([&] {
        im2ht_cuda_forward(at::cuda::getCurrentCUDAStream(),
                    input.data<scalar_t>() ,
                    output.data<scalar_t>(),
                    ht_index.data<scalar_t>(),
                    batch, channel,
                    height, width, 
                    ht_height, ht_width, 
                    num_votes
                    );

    }));
    // std::cout <<"output" <<output.sum() << std::endl;
    // printf("output", output.sum());
    output = output.contiguous().view({batch,channel, ht_height, ht_width});
    return output;
}



at::Tensor
ht_cuda_backward(
            const at::Tensor &grad_output, 
            const at::Tensor &ht_index,
            const int height,
            const int width,
            const int ht_height,
            const int ht_width
            )
{

    AT_ASSERTM(ht_index.is_contiguous(), "ht_index tensor has to be contiguous");
    AT_ASSERTM(grad_output.is_contiguous(), "grad_output tensor has to be contiguous");

    AT_ASSERTM(ht_index.type().is_cuda(), "ht_index must be a CUDA tensor");
    AT_ASSERTM(grad_output.type().is_cuda(), "grad_output must be a CUDA tensor");

    // grad_output: [batch, channel, ht_h, ht_w]
    const int batch = grad_output.size(0);
    const int channel = grad_output.size(1);
    const int ht_height_ = grad_output.size(2);
    const int ht_width_ = grad_output.size(3);

    AT_ASSERTM(ht_height_ == ht_height && ht_width_ == ht_width,
        "given grad_out shape and ht_index shape do not match: (%d x %d vs %d).", ht_height_, ht_width_, ht_height, ht_width);

    const int num_votes = ht_index.size(0);
    const int num_elements = ht_index.size(1);
    AT_ASSERTM(num_elements == 3,
        "ht_index shape != [num_votes x 3]: (%d x %d).", num_votes, num_elements);
    
    auto grad_input = at::zeros({batch, channel, height, width}, grad_output.options());
    
    AT_DISPATCH_FLOATING_TYPES(grad_output.type(), "im2ht_cuda_backward", ([&] {
        im2ht_cuda_backward(at::cuda::getCurrentCUDAStream(),
                    grad_input.data<scalar_t>(),
                    grad_output.data<scalar_t>(),
                    ht_index.data<scalar_t>(),
                    batch, channel,
                    height, width, 
                    ht_height, ht_width,
                    num_votes
                );

    }));

    return grad_input; 
}
